#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2025, wjchen, BSD 3-Clause License
 */
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <vector>
#include "fft_radix2_cuda.h"

namespace learnfft
{
    __global__ void kernelFFTRadix2(double* real_data, double* imag_data, bool forward, int btfly, int len)
    {
        int i = blockIdx.x * blockDim.x + threadIdx.x;
        if (i >= len) return;
        
        int m = len / btfly;
        int step = btfly / 2;
        int k = i % btfly;
        if (k >= step) return;

        int even = i;
        int odd = even + step;

        double arg = (double(k) * double(m) * M_PI * 2.0) / len;
        double _sin = sin(arg);
        double _cos = cos(arg);

        double odd_twiddle_real;
        double odd_twiddle_imag;
        if (forward)
        {
            odd_twiddle_real =
                double(real_data[odd] * _cos + imag_data[odd] * _sin);
            odd_twiddle_imag =
                double(-real_data[odd] * _sin + imag_data[odd] * _cos);
        }
        else
        {
            odd_twiddle_real =
                double(real_data[odd] * _cos - imag_data[odd] * _sin);
            odd_twiddle_imag =
                double(real_data[odd] * _sin + imag_data[odd] * _cos);
        }

        real_data[odd] = real_data[even] - odd_twiddle_real;
        imag_data[odd] = imag_data[even] - odd_twiddle_imag;
        real_data[even] = real_data[even] + odd_twiddle_real;
        imag_data[even] = imag_data[even] + odd_twiddle_imag;
    }

    class FFTCUDAImpl
    {
    public:
        FFTCUDAImpl(size_t size);
        ~FFTCUDAImpl();

        void FFTRadix2CUDA(double* real_data, double* imag_data, bool forward);

    private:
        size_t m_size;
        double *m_dev_real;
        double *m_dev_imag;

    };

    FFTCUDAImpl::FFTCUDAImpl(size_t size) : m_size(size)
    {
        int n_bytes = m_size * sizeof(double);
        hipMalloc((void**)&m_dev_real, n_bytes);
        hipMalloc((void**)&m_dev_imag, n_bytes);
    }

    FFTCUDAImpl::~FFTCUDAImpl() {
        hipFree(m_dev_real);
        hipFree(m_dev_imag);
    }

    void FFTCUDAImpl::FFTRadix2CUDA(double* real_data, double* imag_data, bool forward)
    {

        int n_bytes = m_size * sizeof(double);
        hipMemcpy((void*)m_dev_real, (void*)real_data, n_bytes, hipMemcpyHostToDevice);
        hipMemcpy((void*)m_dev_imag, (void*)imag_data, n_bytes, hipMemcpyHostToDevice);
        int tx = 1024;
        int bx = (m_size + tx - 1) / tx;
        const dim3 blockSize(tx);
        const dim3 gridSize(bx);
        for (int btfly = 2; btfly <= m_size; btfly *= 2)
        {
            kernelFFTRadix2<<<gridSize, blockSize>>>(m_dev_real, m_dev_imag, forward, btfly, m_size);
            hipDeviceSynchronize();
        }
        hipMemcpy((void*)real_data, (void*)m_dev_real, n_bytes, hipMemcpyDeviceToHost);
        hipMemcpy((void*)imag_data, (void*)m_dev_imag, n_bytes, hipMemcpyDeviceToHost);
    }
    
    FFTRadix2CUDA::FFTRadix2CUDA(size_t size)
        : m_size(size), m_bit_reverse_idx(size)
    {
        GenBitReverseOrder(m_size, m_bit_reverse_idx);
        m_impl = new FFTCUDAImpl(size);
    }
     FFTRadix2CUDA::~FFTRadix2CUDA() {
        delete m_impl;
    }

    void FFTRadix2CUDA::Forward(const double* real_in, const double* imag_in, double* real_out, double* imag_out)
    {
        for (int i = 0; i < m_size; ++i)
        {
            real_out[i] = real_in[m_bit_reverse_idx[i]];
            imag_out[i] = imag_in[m_bit_reverse_idx[i]];
        }
        m_impl->FFTRadix2CUDA(real_out, imag_out, true);
    }

    
    void FFTRadix2CUDA::Inverse(const double* real_in, const double* imag_in, double* real_out, double* imag_out)
    {
        for (int i = 0; i < m_size; ++i)
        {
            real_out[i] = real_in[m_bit_reverse_idx[i]];
            imag_out[i] = imag_in[m_bit_reverse_idx[i]];
        }
        m_impl->FFTRadix2CUDA(real_out, imag_out, false);
    }

} // namespace learnfft
